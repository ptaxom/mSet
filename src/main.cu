#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 64

__global__ void matrixMult(const float *a, const float *b, int n, float *c)
{
    int bX = blockIdx.x;
    int bY = blockIdx.y;

    int tX = threadIdx.x;
    int tY = threadIdx.y;

    float sum = 0.0f;

    int indexOfA = n * BLOCK_SIZE * bY + n * tY;
    int indexOfB = BLOCK_SIZE * bX + tX;

    int indexOfC = n * BLOCK_SIZE * bY + BLOCK_SIZE * bX;

    for (int k = 0; k < n; k++)
        sum += a[indexOfA + k] * b[indexOfB + k * n];

    c[indexOfC + n * tY + tX] = sum;
}

void multiplyerGPU(const float *A, const float *B, int N, float *C)
{
    int numBytes = N * N * sizeof(float);

    float *deviceA = nullptr;
    float *deviceB = nullptr;
    float *deviceC = nullptr;

    hipMalloc((void **)&deviceA, numBytes);
    hipMalloc((void **)&deviceB, numBytes);
    hipMalloc((void **)&deviceC, numBytes);

    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocks(N / threads.x, N / threads.y);

    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMemcpy(deviceA, A, numBytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, B, numBytes, hipMemcpyHostToDevice);

    matrixMult<<<blocks, threads>>>(deviceA, deviceB, N, deviceC);

    hipMemcpy(C, deviceC, numBytes, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&gpuTime, start, stop);

    printf("Matrix size = %dx%d\nkernel size = %d\n time spent executing by the GPU: %.2f millseconds\n", N, N, BLOCK_SIZE, gpuTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
}

void printM(float *arr, int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            std::cout << arr[i * N + j] << " ";
        std::cout << std::endl;
    }
    std::cout << "\n\n========\n\n";
}

int main(int, char **)
{
    int N;
    std::cin >> N;
    int size = N * N;
    float *A = new float[size];
    float *B = new float[size];
    float *C = new float[size];
    for (int i = 0; i < size; i++)
    {
        A[i] = (rand() % 1024) / 1024.f;
        B[i] = (rand() % 1024) / 1024.f;
    }
    multiplyerGPU(A, B, N, C);

    return 0;
}
